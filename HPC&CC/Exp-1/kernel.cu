#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>
#include <iostream>
#include <ctime>
using namespace std;

const int MAX_THREAD_NUM = 256;
 // �����СΪ MATRIX_SIZE * MATRIX_SIZE
const int MAX_MATRIX_SIZE = 1024;
// �����鶨��Ϊȫ�֣�������ջ�ڷ������ڴ�
int MATRIX_SIZE;
int THREAD_NUM = 50;
float A[MAX_MATRIX_SIZE * MAX_MATRIX_SIZE], B[MAX_MATRIX_SIZE * MAX_MATRIX_SIZE], C[MAX_MATRIX_SIZE * MAX_MATRIX_SIZE];
float* gpu_A, * gpu_B, * gpu_C;

/* ��ӡGPU�豸��Ϣ */
void printDeviceProps(const hipDeviceProp_t* deviceProp) {

    cout << "�豸�Կ��ͺţ� " << deviceProp->name << endl;
    cout << "�豸ȫ���ڴ���������MBΪ��λ���� " << deviceProp->totalGlobalMem / 1024 / 1024 << endl;
    cout << "�豸��һ���߳̿飨Block���п��õ�������ڴ棨��KBΪ��λ���� " << deviceProp->sharedMemPerBlock / 1024 << endl;
    cout << "�豸��һ���߳̿飨Block���ֿ��õ�32λ�Ĵ��������� " << deviceProp->regsPerBlock << endl;
    cout << "�豸��һ���߳̿飨Block���ɰ���������߳������� " << deviceProp->maxThreadsPerBlock << endl;
    cout << "�豸�ļ��㹦�ܼ���Compute Capability���İ汾�ţ� " << deviceProp->major << "." << deviceProp->minor << endl;
    cout << "�豸�϶ദ������������ " << deviceProp->multiProcessorCount << endl;
}

/* CUDA ��ʼ�� */
bool initCUDA() {
    int count, i;
    hipDeviceProp_t prop;

    hipGetDeviceCount(&count); 

    if (0 == count) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    for (i = 0; i < count; i++) {
        if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if (prop.major >= 1) {
                break;
            }
        }
    }

    if (i == count) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    hipSetDevice(i);

   // printDeviceProps(&prop);

    return true;
}

/* �����ά���飬ʹ��һά����洢 */
void generateMatrix(float* mat, int size) {
    int i, j;
    for (i = 0; i < size; i++) {
        for (j = 0; j < size; j++) {
            mat[i * size + j] = rand() %100;
        }
    }
}

/* ��ӡ���� */
void printMatrix(float* mat, int size) {
    int i, j;
    for (i = 0; i < size; i++) {
        for (j = 0; j < size; j++) {
            printf("%f ", mat[i * size + j]);
        }
        printf("\n");
    }
    printf("\n");
}

// kernel������ʵ�־���˷�
__global__ static void matrixMultiplication(const float* A, const float* B, float* C, int size, int threadnum) {
    // ��ǰ�߳�����Block�ı�ţ���0��ʼ��
    const int block_id = blockIdx.x;
    //�����̱߳�ţ���0��ʼ��
    const int thread_id = threadIdx.x;
    int i;
    int index, row, column;
    float s;

    // ��ǰ�߳�ȫ����������Block�ڵ��߳�������
    index = block_id * threadnum + thread_id;

    /* ��ǰ�߳̽�����C[row][column] */
    row = index / size;
    column = index % size;

    s = 0.0f;
    if (row < size && column < size) {
        // A[row][0], A[row][1], A[row][2] ... A[row][size]
        // B[0]column], B[1][column], B[2][column] ... B[size][column]
        for (i = 0; i < size; i++) {
            s += A[row * size + i] * B[i * size + column];
        }
        C[row * size + column] = s;
    }
}

void serial(const float* a, const float* b, float* c, int size) {
    memset(c, 0, sizeof(c));
    int i, j, k;
    for (i = 0; i < size; i++) {
        for (j = 0; j < size; j++) {
            for (k = 0; k < size; k++) {
                c[i * size + j] += a[i * size + k] * b[k * size + j];
            }
        }
    }
}
int main(void) {

    FILE* file = fopen("data.dat", "w");
    if (!initCUDA()) {
        return 0;
    }
    const int tot = 1024;
    for (int cnt = 0; cnt < tot; cnt++) {

        THREAD_NUM = 1 + (1 + rand()) % MAX_THREAD_NUM;
        MATRIX_SIZE = 1 + (THREAD_NUM + rand()) % MAX_MATRIX_SIZE;
        system("cls");
        printf("[ %d / %d ] \n",1+cnt,tot);
        const int block_num = (MATRIX_SIZE * MATRIX_SIZE + THREAD_NUM - 1) / THREAD_NUM;
        int i, j;
    
        /* �����¼�����¼ʱ��*/
        hipEvent_t start, stop ;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        /* �������� */
        generateMatrix(A, MATRIX_SIZE);
        generateMatrix(B, MATRIX_SIZE);

        /* �����Դ� */
        hipMalloc((void**)&gpu_A, sizeof(float) * MATRIX_SIZE * MATRIX_SIZE);
        hipMalloc((void**)&gpu_B, sizeof(float) * MATRIX_SIZE * MATRIX_SIZE);
        hipMalloc((void**)&gpu_C, sizeof(float) * MATRIX_SIZE * MATRIX_SIZE);

        /* ����������ڴ濽�����Դ� */
        hipMemcpy(gpu_A, A, sizeof(float) * MATRIX_SIZE * MATRIX_SIZE, hipMemcpyHostToDevice);
        hipMemcpy(gpu_B, B, sizeof(float) * MATRIX_SIZE * MATRIX_SIZE, hipMemcpyHostToDevice);

        // ִ��kernel����
        matrixMultiplication <<< block_num, THREAD_NUM, 0 >>> (gpu_A, gpu_B, gpu_C, MATRIX_SIZE, THREAD_NUM);

        // ��������Դ濽�������ڴ�
        hipMemcpy(C, gpu_C, sizeof(float) * MATRIX_SIZE * MATRIX_SIZE, hipMemcpyDeviceToHost);

        /* �ͷ��Դ�ռ� */
        hipFree(gpu_A);
        hipFree(gpu_B);
        hipFree(gpu_C);

        /* ��¼����ʱ�� */
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float elapsedTime;
        hipEventElapsedTime(&elapsedTime, start, stop);


        

        //printf("Matrix size: [%d * %d] * [%d * %d] \n ", MATRIX_SIZE, MATRIX_SIZE, MATRIX_SIZE, MATRIX_SIZE);
        //printf("Time used: %f ms\n ",elapsedTime);

        /* Optional */
        //printMatrix(A, MATRIX_SIZE);
        //printMatrix(B, MATRIX_SIZE);
        //printMatrix(C, MATRIX_SIZE);
    

        /* Serial Matrix Multiply Algo */
    
        
    
        double SerialStart, SerialEnd;
        SerialStart = (double)clock();
        //cout << SerialStart << endl;
        serial(A, B, C, MATRIX_SIZE);
        SerialEnd = (double)clock();
        printf("Time used: %lf ms\n ", SerialEnd-SerialStart);
    
        fprintf(file, "%d %d %d %f %f \n", cnt, THREAD_NUM, MATRIX_SIZE, elapsedTime, SerialEnd - SerialStart);
    
    }
    
    system("pause");
    // return 0;
}